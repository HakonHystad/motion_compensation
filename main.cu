
//////////////////////////////////////////////////////////////
// dependencies
/////////////////////////////////////////////////////////////

#include <iostream>
#include <string>

#include "filter/settings.cuh"
#include "filter//utilities.cuh"
#include "filter/filter.cuh"

#define _TEST_FILTER_

#ifdef _TEST_FILTER_

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;

#endif



int main(int argc, char *argv[])
{
    //////////////////////////////////////////////////////////////
    // set initial values for the filter 
    /////////////////////////////////////////////////////////////
    
    float camera1[12];
    float camera2[12];
    float worldPoints[N_WPOINTS*3];

    // read config file
    if (!getConfig( camera1, camera2, worldPoints) )
	exit( EXIT_FAILURE );

    // x,y,z, x',y',z',a,b,g,a',b',g'
    float initialStates[N_STATES] = {1.6, -1.5, 2, 0.0324, 0, 0,\
				    0, 0, 0, 0, 0, 0};
    // 0.1m, 0.1m/s, ~5.7 deg, ~5.7deg/s
    float initialSigma[N_STATES] = {0.1, 0.1, 0.1, 0.1, 0.1, 0.1,\
				    0.1, 0.1, 0.1, 0.1, 0.1, 0.1};

    /* GPU DATA */
    float *d_camera1;
    float *d_camera2;

    hipMalloc( &d_camera1, sizeof(camera1) );
    hipMalloc( &d_camera2, sizeof(camera2) );
    checkCUDAError("malloc main");

    hipMemcpy( d_camera1, camera1, sizeof( camera1 ), hipMemcpyHostToDevice );
    hipMemcpy( d_camera2, camera2, sizeof( camera1 ), hipMemcpyHostToDevice );
    checkCUDAError("memcpy main");

    //////////////////////////////////////////////////////////////
    // set up image loading
    /////////////////////////////////////////////////////////////
    uchar *image;
    #ifdef _TEST_FILTER_
    cv::Mat im = cv::imread( "./data/seq_500.png", CV_LOAD_IMAGE_GRAYSCALE );
    image = im.data;
    #endif
    /* end testing */
    hipHostMalloc( (void**)&image, IM_W*IM_H*sizeof(uchar) );

    
    hipArray *cuArray;
    // Allocate CUDA array in device memory
    auto channelDesc = hipCreateChannelDesc<uchar>();
    hipMallocArray(&cuArray, &channelDesc, IM_W, IM_H);
    checkCUDAError("malloc array");
    auto texObj = makeTexture( cuArray );


    //////////////////////////////////////////////////////////////
    // make filter instance
    /////////////////////////////////////////////////////////////

    auto sir = Filter( initialStates, initialSigma, worldPoints );

    //////////////////////////////////////////////////////////////
    // get currect image from buffer
    /////////////////////////////////////////////////////////////

    // TODO: image acquisition
    // image = something
    
    float *camera = d_camera1;// TODO: set which camera took the picture
    
    // copy to GPU, TODO: asynchrounous
    
    #ifdef _TEST_FILTER_
    std::cout << "=======================================\n";
	
    std::cout << "Timings for " <<  N_PARTICLES << " particles\n";
    std::cout << "---------------------------------------\n";	

    double start, end, total=0;
    
    start = get_current_time();
    #endif
    
    hipMemcpyToArray(cuArray, 0, 0, image, IM_W*IM_H*sizeof(uchar) , hipMemcpyHostToDevice);
    checkCUDAError("mempcy texture");
	
    // TODO: get timestamp
    float prevTime = 0;
    float newTime = 1.0f/60;

    //////////////////////////////////////////////////////////////
    // perform filtering 
    /////////////////////////////////////////////////////////////

    
    sir.update(prevTime, newTime, camera, texObj );

    prevTime = newTime;

    sir.resample();

    sir.mean();

    hipDeviceSynchronize();

    #ifdef _TEST_FILTER_
    end = get_current_time();
    total += end-start;
    
    std::cout << "---------------------------------------\n";	
    std::cout << "Total time: " << total << "s\n";
    #endif

    //////////////////////////////////////////////////////////////
    // process new states 
    /////////////////////////////////////////////////////////////
    // TODO: prediction + smoothing?

    // TODO: send pose to robot as x,y,z in mm? and A,B,C in kuka euler coordinates


    // TODO: repeat with new image



    //////////////////////////////////////////////////////////////
    // clean up 
    /////////////////////////////////////////////////////////////


    hipFree( d_camera1 );
    hipFree( d_camera2 );
    hipFreeArray( cuArray );
    hipDestroyTextureObject(texObj);



    return 0;
}


