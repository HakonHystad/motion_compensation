#include "hip/hip_runtime.h"

#define MAX_FPS 40
#define _TEST_FILTER_
//#define _WITH_ROBOT_

//////////////////////////////////////////////////////////////
// dependencies
/////////////////////////////////////////////////////////////

#include <iostream>
#include <string>
#include <unistd.h>
#include <mutex>

#include "../filter/settings.cuh"
#include "../filter//utilities.cuh"
#include "../filter/filter.cuh"

#include "../robot/robot.h"
#include "../camera/continousCamera.h"



int main(int argc, char *argv[])
{
    //////////////////////////////////////////////////////////////
    // set initial values for the filter 
    /////////////////////////////////////////////////////////////
    
    float camera1[12];
    float camera2[12];
    float worldPoints[N_WPOINTS*3];

    std::string camID1;
    std::string camID2;

    // read config file
    if (!getConfig( camera1, camera2, worldPoints, camID1, camID2) )
	exit( EXIT_FAILURE );

#ifdef _WITH_ROBOT_
    float initialStates[N_STATES] = {1.5, 0.92, 2.0, 0.0324, 0, 0,	\
				    3.1415, 0, 0, 0, 0, 0};
#else
    float initialStates[N_STATES] = {1.5, 0, 2.0, 0.0324, 0, 0,	\
				    3.1415, 0, 0, 0, 0, 0};
#endif
    // 0.1m, 0.01m/s, ~5.7 deg, ~5.7deg/s
    float initialSigma[N_STATES] = {0.1, 0.1, 0.1, 0.01, 0.01, 0.01,\
				    0.1, 0.1, 0.1, 0.1, 0.1, 0.1};

    /* GPU DATA */
    float *d_camera1;
    float *d_camera2;

    hipMalloc( &d_camera1, sizeof(camera1) );
    hipMalloc( &d_camera2, sizeof(camera2) );
    checkCUDAError("malloc main");

    hipMemcpy( d_camera1, camera1, sizeof( camera1 ), hipMemcpyHostToDevice );
    hipMemcpy( d_camera2, camera2, sizeof( camera1 ), hipMemcpyHostToDevice );
    checkCUDAError("memcpy main");

    //////////////////////////////////////////////////////////////
    // set up image loading
    /////////////////////////////////////////////////////////////
    uchar *image = NULL;
    uchar *image1;
    uchar *image2;
    
    // make pinned memory
    hipHostMalloc( (void**)&image1, IM_W*IM_H*sizeof(uchar) );
    hipHostMalloc( (void**)&image2, IM_W*IM_H*sizeof(uchar) );
    
    hipArray *cuArray;
    // Allocate CUDA array in device memory
    auto channelDesc = hipCreateChannelDesc<uchar>();
    hipMallocArray(&cuArray, &channelDesc, IM_W, IM_H);
    checkCUDAError("malloc array");
    auto texObj = makeTexture( cuArray );

    // put image transfer and motion kernel in separate kernels to get async transfer
    hipStream_t memStream = 0;
    hipStream_t motionStream = 0;
    // comment out to run in default stream
    hipStreamCreate(&memStream);
    hipStreamCreate(&motionStream);
    checkCUDAError("make stream");

    HH::ContinousCamera *cam = NULL;
    unsigned long long prevTimestamp=0;
    std::atomic<unsigned long long> newTimestamp(0);
    
    std::mutex mtx;
    std::unique_lock<std::mutex> lock(mtx);
    lock.unlock();
    
    
    try
    {
      //      cam = new HH::Camera( image, IM_H*IM_W, "02-2165A-07078", "02-2165A-07077" );
	cam = new HH::ContinousCamera( image1, image2, IM_H*IM_W, mtx, newTimestamp, image, camID1, camID2 );
    }catch(VmbErrorType)
    {
	exit( EXIT_FAILURE );
    }
    // sync via precission time protocol
    if( !cam->startPTP() )
	exit( EXIT_FAILURE );



    //////////////////////////////////////////////////////////////
    // create instances
    /////////////////////////////////////////////////////////////

    auto sir = Filter( initialStates, initialSigma, worldPoints );

#ifdef _WITH_ROBOT_
    HH::Robot robot;

    for (int i = 0; i < 3; ++i)
	robot.pose[i] = initialStates[i];
    for (int i = 0; i < 3; ++i)
	robot.pose[i+3] = initialStates[i+ALPHA_IDX];

	std::cout << "Initiating at ";
	for( int i = 0; i<6; i++ )
	     std::cout << robot.pose[i] << " ";
	std::cout << std::endl;

    // move to start position
    robot.move(true);
    // wait for it to finish
    while( !robot.poseReached() )
	sleep( 0.2 );
#endif
	
    //////////////////////////////////////////////////////////////
    // filter loop
    /////////////////////////////////////////////////////////////
    double pose[6];
    std::ofstream fd_pose("./data/measured_poses.txt", std::ios::trunc );

    if( !fd_pose.is_open() )
    {
	std::cerr << "Could not open pose\n";
	exit( EXIT_FAILURE );
    }

    std::ofstream fd_calc_pose("./data/calculated_poses.txt", std::ios::trunc );
    if( !fd_calc_pose.is_open() )
    {
	std::cerr << "Could not open calc pose\n";
	exit( EXIT_FAILURE );
    }

    
    float *camera = d_camera1;
    
    cam->startCapture();// start continous capture
    float prevTime = 0;
    float newTime = 1.0f/60;

    std::cout << "starting movement\n";

#ifdef _WITH_ROBOT_
    robot.pose[1] -= 2*initialStates[1];
    robot.move(true);
#endif

    // use first image to set initial timestamp.. kind of a hack
    
    while( prevTimestamp >= newTimestamp )
    {
	// loop until new frame is in
    }
    prevTimestamp = newTimestamp;
    prevTime = (float)prevTimestamp/1e9;

#ifdef _WITH_ROBOT_
    while(!robot.poseReached())
    {
#else
	while(true)
	{
#endif
	
#ifdef _TEST_FILTER_
	std::cout << "=======================================\n";
	
	std::cout << "Timings for " <<  N_PARTICLES << " particles\n";
	std::cout << "---------------------------------------\n";	

	double start, end, total=0;
    
	start = get_current_time();
#endif


    
	//////////////////////////////////////////////////////////////
	// get currect image from buffer
	/////////////////////////////////////////////////////////////

	
	while( prevTimestamp >= newTimestamp )
	{
	    // loop until new frame is in
	}
	
	lock.lock();
	
	// maybe should not be async b.c of mutex..
	cudaMemcpyToArrayAsync(cuArray, 0, 0, image, IM_W*IM_H*sizeof(uchar) , hipMemcpyHostToDevice,  memStream);
	checkCUDAError("mempcy texture");

	
	prevTimestamp = newTimestamp;

	if( image==image1 )
	    camera = d_camera1;
	else if( image==image2 )
	    camera = d_camera2;
	else
	{
	    std::cerr << "Not expected image buffer\n";
	    break;
	}
	
	lock.unlock();
	
	newTime = (float)prevTimestamp/1e9;

	
#ifdef _WITH_ROBOT_
	// save the measured pose
	robot.getPose( pose );
	for (int j = 0; j < 6; ++j)
	    fd_pose << pose[j] << " ";
	fd_pose << std::endl;
#endif
	
	//////////////////////////////////////////////////////////////
	// perform filtering 
	/////////////////////////////////////////////////////////////

    
	sir.update(prevTime, newTime, camera, texObj, motionStream );
	prevTime = newTime;

	prevTime = newTime;

	sir.resample();

	sir.mean();

	hipDeviceSynchronize();

	//////////////////////////////////////////////////////////////
	// pass on estimated states
	/////////////////////////////////////////////////////////////


	for (int i = 0; i < N_STATES; ++i)
	    fd_calc_pose << sir[i] << " ";
//	for (int i = 0; i < 3; ++i)
//	    fd_calc_pose << sir[i+ALPHA_IDX] << " ";
	fd_calc_pose << std::endl;


#ifdef _TEST_FILTER_
	end = get_current_time();
	total += end-start;
    
	std::cout << "---------------------------------------\n";	
	std::cout << "Total time: " << total << "s\n";
	std::cout << "States: ";
	for (int i = 0; i < N_STATES; ++i)
	    std::cout << sir[i] << " ";
	std::cout << std::endl;
#endif
    }

    cam->stopCapture();


    //////////////////////////////////////////////////////////////
    // clean up 
    /////////////////////////////////////////////////////////////


    hipFree( d_camera1 );
    hipFree( d_camera2 );
    hipFreeArray( cuArray );
    hipDestroyTextureObject(texObj);
    hipStreamDestroy(memStream);
    hipStreamDestroy(motionStream);
    hipHostFree(image1);
    hipHostFree(image2);
    
    delete cam;

    return 0;
}

