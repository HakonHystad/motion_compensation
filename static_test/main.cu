#include "hip/hip_runtime.h"

#define MAX_FPS 40
#define _TEST_FILTER_
//#define _WITH_ROBOT_

//////////////////////////////////////////////////////////////
// dependencies
/////////////////////////////////////////////////////////////

#include <iostream>
#include <string>
#include <unistd.h>
#include <mutex>

#include "../filter/settings.cuh"
#include "../filter//utilities.cuh"
#include "../filter/filter.cuh"

#include "../robot/robot.h"
#include "../camera/continousCamera.h"



int main(int argc, char *argv[])
{
    //////////////////////////////////////////////////////////////
    // set initial values for the filter 
    /////////////////////////////////////////////////////////////
    
    float camera1[12];
    float camera2[12];
    float worldPoints[N_WPOINTS*3];

    std::string camID1;
    std::string camID2;

    // read config file
    if (!getConfig( camera1, camera2, worldPoints, camID1, camID2) )
	exit( EXIT_FAILURE );

#ifdef _WITH_ROBOT_
    float initialStates[N_STATES] = {1.5, 0.92, 2.0, 0.0324, 0, 0,	\
				    0, 0, 0, 0, 0, 0};
#else
    float initialStates[N_STATES] = {1.5, 0, 1.9, 0, 0, 0,	\
				    0, 0, 0, 0, 0, 0};
#endif
    // 0.1m, 0.01m/s, ~5.7 deg, ~5.7deg/s
    float initialSigma[N_STATES] = {0.1, 0.1, 0.1, 0.01, 0.01, 0.01,\
				    0.1, 0.1, 0.1, 0.1, 0.1, 0.1};

    /* GPU DATA */
    float *d_camera1;
    float *d_camera2;

    hipMalloc( &d_camera1, sizeof(camera1) );
    hipMalloc( &d_camera2, sizeof(camera2) );
    checkCUDAError("malloc main");

    hipMemcpy( d_camera1, camera1, sizeof( camera1 ), hipMemcpyHostToDevice );
    hipMemcpy( d_camera2, camera2, sizeof( camera1 ), hipMemcpyHostToDevice );
    checkCUDAError("memcpy main");

    //////////////////////////////////////////////////////////////
    // set up image loading
    /////////////////////////////////////////////////////////////
    uchar *image;
    uchar *image1;
    uchar *image2;
    
    // make pinned memory
    hipHostMalloc( (void**)&image1, IM_W*IM_H*sizeof(uchar) );
    hipHostMalloc( (void**)&image2, IM_W*IM_H*sizeof(uchar) );

//    std::cout << "image 1: " << (void*)image1 << "\nimage 2: " << (void*)image2 << std::endl; 
    
    hipArray *cuArray;
    // Allocate CUDA array in device memory
    auto channelDesc = hipCreateChannelDesc<uchar>();
    hipMallocArray(&cuArray, &channelDesc, IM_W, IM_H);
    checkCUDAError("malloc array");
    auto texObj = makeTexture( cuArray );

    // put image transfer and motion kernel in separate kernels to get async transfer
    hipStream_t memStream = 0;
    hipStream_t motionStream = 0;
    // comment out to run in default stream
    hipStreamCreate(&memStream);
    hipStreamCreate(&motionStream);
    checkCUDAError("make stream");

    HH::ContinousCamera *cam = NULL;
    unsigned long long prevTimestamp=0;
    std::atomic<unsigned long long> newTimestamp(0);
    std::atomic<int> currentCam(1);
    
    std::mutex mtx;
    std::unique_lock<std::mutex> lock(mtx);
    lock.unlock();
    
    
    try
    {
      //      cam = new HH::Camera( image, IM_H*IM_W, "02-2165A-07078", "02-2165A-07077" );
	cam = new HH::ContinousCamera( image1, image2, IM_H*IM_W, mtx, newTimestamp, currentCam, camID1, camID2 );
    }catch(VmbErrorType)
    {
	exit( EXIT_FAILURE );
    }
    // sync via precission time protocol
    if( !cam->startPTP() )
	exit( EXIT_FAILURE );



    //////////////////////////////////////////////////////////////
    // create instances
    /////////////////////////////////////////////////////////////

    auto sir = Filter( initialStates, initialSigma, worldPoints );

#ifdef _WITH_ROBOT_
    HH::Robot robot;

    for (int i = 0; i < 3; ++i)
	robot.pose[i] = initialStates[i];
    for (int i = 0; i < 3; ++i)
	robot.pose[i+3] = initialStates[i+ALPHA_IDX];

	std::cout << "Initiating at ";
	for( int i = 0; i<6; i++ )
	     std::cout << robot.pose[i] << " ";
	std::cout << std::endl;

    // move to start position
    robot.move(true);
    // wait for it to finish
    while( !robot.poseReached() )
	sleep( 0.2 );
#endif
	
    //////////////////////////////////////////////////////////////
    // filter loop
    /////////////////////////////////////////////////////////////
    double pose[6];
    std::ofstream fd_pose("./data/measured_poses.txt", std::ios::trunc );

    if( !fd_pose.is_open() )
    {
	std::cerr << "Could not open pose\n";
	exit( EXIT_FAILURE );
    }

    std::ofstream fd_calc_pose("./data/calculated_poses.txt", std::ios::trunc );
    if( !fd_calc_pose.is_open() )
    {
	std::cerr << "Could not open calc pose\n";
	exit( EXIT_FAILURE );
    }

    std::ofstream fd_timing("./data/timings.txt", std::ios::trunc );

    if( !fd_timing.is_open() )
    {
	std::cerr << "Could not open timings\n";
	exit( EXIT_FAILURE );
    }

    std::cout << "Starting capture" << std::endl;
    
    float *camera = d_camera1;
    
    cam->startCapture();// start continous capture
    float prevTime = 0;
    float newTime = 1.0f/60;

    std::cout << "starting movement\n";

#ifdef _WITH_ROBOT_
    robot.pose[1] -= 2*initialStates[1];
    robot.move(true);
#endif

    // use first image to set initial timestamp.. kind of a hack
    
    while( prevTimestamp >= newTimestamp )
    {
	// loop until new frame is in
	//std::cout << (float)newTimestamp/1e9 << std::endl;
      //std::cout << (newTimestamp - prevTimestamp)/1e9 << std::endl;
    }
    prevTimestamp = newTimestamp;
    prevTime = (float)prevTimestamp/1e9;


    //    cam->stopCapture();

#ifdef _WITH_ROBOT_
    while(!robot.poseReached())
    {
#else
	while(true)
	{
#endif
	
#ifdef _TEST_FILTER_
	std::cout << "=======================================\n";
	
	std::cout << "Timings for " <<  N_PARTICLES << " particles\n";
	std::cout << "---------------------------------------\n";	

	double start, end, total=0;
    
	start = get_current_time();
#endif


    
	//////////////////////////////////////////////////////////////
	// get currect image from buffer
	/////////////////////////////////////////////////////////////
	/*
	if( currentCam == 1 )
	  cam->startCapture(2);
	else
	  cam->startCapture(1);
	*/

	while( (newTimestamp - prevTimestamp)<(0.005*1e9) )// require frames to be at least 0.5ms between eachother
	{
	    // loop until new frame is in
	    //std::cout << (float)newTimestamp/1e9 << std::endl;
//	  std::cout << (newTimestamp - prevTimestamp)/1e9 << std::endl;
	}
	
	lock.lock();

	if( currentCam == 1 )
	  {
	    camera = d_camera1;
	    image = image1;
	  }	  
	else if( currentCam == 2 )
	  {
	    camera = d_camera2;
	    image = image2;
	  }
	else
	{
	    std::cerr << "Not expected currentCam\n";
	    lock.unlock();
	    break;
	}

	
	// maybe should not be async b.c of mutex..
//	std::cout << "Address: " << (void*)image << std::endl;

//	cudaMemcpyToArrayAsync(cuArray, 0, 0, image, IM_W*IM_H*sizeof(uchar) , hipMemcpyHostToDevice,  memStream);
	hipMemcpyToArray(cuArray,0,0,image,IM_W*IM_H*sizeof(uchar), hipMemcpyHostToDevice );
	checkCUDAError("mempcy texture");

	//cam->stopCapture(currentCam);
	prevTimestamp = newTimestamp;
	std::cout << "Using camera " << currentCam << ": " << (void*)image << std::endl;
	
	lock.unlock();
	
	newTime = (float)prevTimestamp/1e9;

	std::cout << "Integrated over " << newTime - prevTime << "s" << std::endl;
	// save time
	fd_timing << newTime - prevTime << std::endl;


	
#ifdef _WITH_ROBOT_
	// save the measured pose
	robot.getPose( pose );
	for (int j = 0; j < 6; ++j)
	    fd_pose << pose[j] << " ";
	fd_pose << std::endl;
#endif
	
	//////////////////////////////////////////////////////////////
	// perform filtering 
	/////////////////////////////////////////////////////////////

    
	sir.update(prevTime, newTime, camera, texObj, motionStream );
	prevTime = newTime;


	sir.resample();

	sir.mean();

	hipDeviceSynchronize();

	//////////////////////////////////////////////////////////////
	// pass on estimated states
	/////////////////////////////////////////////////////////////


	for (int i = 0; i < N_STATES; ++i)
	    fd_calc_pose << sir[i] << " ";
//	for (int i = 0; i < 3; ++i)
//	    fd_calc_pose << sir[i+ALPHA_IDX] << " ";
	fd_calc_pose << std::endl;


#ifdef _TEST_FILTER_
	end = get_current_time();
	total += end-start;
    
	std::cout << "---------------------------------------\n";	
	std::cout << "Total time: " << total << "s\n";
	std::cout << "States: ";
	for (int i = 0; i < N_STATES; ++i)
	    std::cout << sir[i] << " ";
	std::cout << std::endl;
#endif

    }

    cam->stopCapture();


    //////////////////////////////////////////////////////////////
    // clean up 
    /////////////////////////////////////////////////////////////


    hipFree( d_camera1 );
    hipFree( d_camera2 );
    hipFreeArray( cuArray );
    hipDestroyTextureObject(texObj);
    hipStreamDestroy(memStream);
    hipStreamDestroy(motionStream);
    hipHostFree(image1);
    hipHostFree(image2);

    cam->shutdown();
    delete cam;

    return 0;
}

